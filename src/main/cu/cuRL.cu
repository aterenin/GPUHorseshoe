
#include <hip/hip_runtime.h>
/*
 *  Copyright 2016 Alexander Terenin
 *
 *  Licensed under the Apache License, Version 2.0 (the "License")
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *  http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 * /
 */



/*
 * Function         : cuda_RL
 * Purpose          : adds sqrt(L) to lower diagonal of 2*n*n matrix R
 * Argument n       : size of L, R
 * Argument *L      : pointer to L vector
 * Argument *R      : pointer to R matrix
 * Output           : mutates R and stores result in its place
 */
extern "C"
__global__ void cuda_RL(int n, float *L, float* R) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i < n)
    R[(2*i+1)*n + i] = sqrtf(L[i]);
}