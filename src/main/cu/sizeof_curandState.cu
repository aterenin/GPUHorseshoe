/*
 *  Copyright 2016 Alexander Terenin
 *
 *  Licensed under the Apache License, Version 2.0 (the "License")
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *  http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 * /
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// utility script to print sizeof(curandState), which is nowhere to be found in JCuda
int main() {
    hiprandState *states;
    hiprandStatePhilox4_32_10_t *philox;
    hiprandStateMRG32k3a *mrg;
    hipMalloc((void **)&states, 64 * 64 * sizeof(hiprandState));
    hipMalloc((void **)&philox, 64 * 64 * sizeof(hiprandStatePhilox4_32_10_t));
    hipMalloc((void **)&mrg, 64 * 64 * sizeof(hiprandStateMRG32k3a));
    printf("sizeof(curandState) %lu\n",sizeof(hiprandState));
    printf("sizeof(curandStatePhilox4_32_10_t) %lu\n",sizeof(hiprandStatePhilox4_32_10_t));
    printf("sizeof(curandStateMRG32k3a) %lu\n",sizeof(hiprandStateMRG32k3a));
}


